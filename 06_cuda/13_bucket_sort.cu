#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <vector>


__global__ void count_keys(int *gbucket, int *gkey, int n){
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) atomicAdd(&gbucket[gkey[i]], 1);
} 

__global__ void sum_keys(int *gsum, int *gbucket, int range){
  if (threadIdx.x==0){
    for(int i = blockIdx.x; i < range;i++){
      atomicAdd(&gsum[i], gbucket[blockIdx.x]);
    }
  }
}

__global__ void set(int *gsum, int *gkey, int range, int n){
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n){
    for(int ki = 0; ki < range; ki++){
      if (i < gsum[ki]){
        gkey[i] = ki;
	break;
      }
    }
  }
}


int main() {
  int n = 50;
  int range = 5;
//  std::vector<int> key(n);
  int key[50];
  for (int i=0; i<n; i++) {
    key[i] = rand() % range;
    printf("%d ",key[i]);
  }
  printf("\n");

  int blocks = range;
  int kbsize = (n + blocks - 1) / blocks;
  int *gkey, *gbucket, *gsum;

  hipMallocManaged(&gkey   , n*sizeof(int));
  hipMallocManaged(&gbucket, range*sizeof(int));
  hipMallocManaged(&gsum   , range*sizeof(int));
  hipMemcpy(gkey, key, n*sizeof(int), hipMemcpyDeviceToHost);

  // initialize
  hipMemset(gbucket, 0, range*sizeof(int));
  hipMemset(gsum, 0, range*sizeof(int));

  // counting
  count_keys<<<blocks, kbsize>>>(gbucket, gkey, n);
  hipDeviceSynchronize();
  
  // sum
  sum_keys<<<blocks, 1>>>(gsum, gbucket, range);
  hipDeviceSynchronize();

  // set
  set<<<kbsize, blocks>>>(gsum, gkey, range, n);
  hipDeviceSynchronize();
  
  hipMemcpy(key, gkey, n*sizeof(int), hipMemcpyDeviceToHost);

  for (int i=0; i<n; i++) {
    printf("%d ",key[i]);
  }
  printf("\n");
  hipFree(gkey);
  hipFree(gbucket);
  hipFree(gsum);
}
